#include "hip/hip_runtime.h"
#include <iostream>
#include <limits.h>
#include <stdlib.h>
#include <ctime>
#include <sstream>
#include <string>
#include <math.h>
#include "test_map.hpp"
#include "gpu_hashtable.hpp"


using namespace std;
/**
 * Funtion hash
 */
__device__ unsigned int fHash(int key) {
	return (unsigned long long)key *  110477914016779llu % 452517535812813007llu;
}
/**
 *Function constructor GpuHashTable
 */
GpuHashTable::GpuHashTable(int size) {
	glbGpuAllocator->_cudaMallocManaged((void **) &GPUHTable.data, size * sizeof(pair<int, int>));
	hipMemset(GPUHTable.data, 0, size * sizeof(pair<int, int>));
	GPUHTable.count = 0;
	GPUHTable.size = size;	
}
/**
 * Function desctructor GpuHashTable
 */
GpuHashTable::~GpuHashTable() {
	glbGpuAllocator->_cudaFree(GPUHTable.data);
	GPUHTable.count = GPUHTable.size = 0;	
}
/**
 * Kernel funtion reshape
 */
__global__ void kernel_reshape(pair<int, int> *oldData, pair<int, int> *newData, int oldSize, int newSize) {
	int  i = 0, idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < oldSize && oldData[idx].first != 0) { // replace the old key:value to new position
		while (i < oldSize) { // search liniar free slot
			if (atomicCAS(&newData[(fHash(oldData[idx].first) + i) % newSize].first, 0, oldData[idx].first) == 0) {
				atomicExch(&newData[(fHash(oldData[idx].first) + i) % newSize].second, oldData[idx].second);
				return;
			}
			i++;
		}
	}
}
/**
 * Function reshape
 * Performs resize of the hashtable based on load factor
 */
void GpuHashTable::reshape(int numBucketsReshape) {
	pair<int, int> *data; // allocation of memory
	glbGpuAllocator->_cudaMallocManaged((void **) &data, numBucketsReshape * sizeof(pair<int, int>));
	hipMemset(data, 0, numBucketsReshape * sizeof(pair<int, int>));
	// call kernel
	unsigned int blocks = ceil(1.0 * numBucketsReshape / THREADS_PER_BLOCK);
	kernel_reshape<<<blocks, THREADS_PER_BLOCK>>>(GPUHTable.data, data, GPUHTable.size, numBucketsReshape);
	hipDeviceSynchronize(); // wait
	cudaCheckError(); // ceck error
	GPUHTable.size = numBucketsReshape; // update size
	glbGpuAllocator->_cudaFree(GPUHTable.data);
	GPUHTable.data = data; // update data
}
/**
 * Kernel funtion insertBatch
 */
__global__ void kernel_insert(int *keys, int *values, int numKeys, Htable GPUHTable) {
	int key, i = 0, idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < numKeys) { // insert key:value
		while (i < GPUHTable.size) { // search liniar free slot
			key = atomicCAS(&(GPUHTable.data[(fHash(keys[idx]) + i) % GPUHTable.size].first), 0, keys[idx]);
			if (key == 0) { // free slot
				atomicExch(&(GPUHTable.data[(fHash(keys[idx]) + i) % GPUHTable.size].second), values[idx]);
				return;
			} else if (key == keys[idx]) { // update
				atomicExch(&(GPUHTable.data[(fHash(keys[idx]) + i) % GPUHTable.size].second), values[idx]);
				GPUHTable.count--;
				return;
			}
			i++;
		}
	}
}
/**
 * Function insertBatch
 * Inserts a batch of key:value, using GPU and wrapper allocators
 */
bool GpuHashTable::insertBatch(int *keys, int *values, int numKeys) {
	int *deviceKeys, *deviceValues; // allocation of memory
	glbGpuAllocator->_cudaMalloc((void **)&deviceKeys, numKeys * sizeof(int));
	glbGpuAllocator->_cudaMalloc((void **)&deviceValues, numKeys * sizeof(int));

	if (!deviceKeys || !deviceValues) { // ceck memory
		return false;
	}
	if ((GPUHTable.count + numKeys) >= GPUHTable.size) // ceck load factor
		reshape((GPUHTable.count + numKeys) / MIN_LOAD);

	hipMemcpy(deviceKeys, keys, numKeys * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(deviceValues, values, numKeys * sizeof(int), hipMemcpyHostToDevice);
	// call kernel
	unsigned int blocks = ceil(1.0 * numKeys / THREADS_PER_BLOCK);
	kernel_insert<<<blocks, THREADS_PER_BLOCK>>>(deviceKeys, deviceValues, numKeys, GPUHTable);
	hipDeviceSynchronize(); // wait
	cudaCheckError(); // ceck error
	GPUHTable.count += numKeys; // update count
	glbGpuAllocator->_cudaFree(deviceKeys);// free of memory
	glbGpuAllocator->_cudaFree(deviceValues);
	return true;
}
/**
 * Kernel function getBatch
 */
__global__ void kernel_get(int *keys, int *values, int numKeys, Htable GPUHTable) {
	int i = 0, idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < numKeys) { // find value for key
		while(i < GPUHTable.size) { // find liniar key
			if (GPUHTable.data[(fHash(keys[idx]) + i) % GPUHTable.size].first == keys[idx]) {
				atomicExch(&values[idx], GPUHTable.data[(fHash(keys[idx]) + i) % GPUHTable.size].second);
				return;
			}
			i++;
		}
		values[idx] = -1;
	}
}
/**
 * Function getBatch
 * Gets a batch of key:value, using GPU
 */
int* GpuHashTable::getBatch(int* keys, int numKeys) {
	int *deviceKeys, *deviceValues, *hostValues; // alocation of memory
	glbGpuAllocator->_cudaMalloc((void **)&deviceKeys, numKeys * sizeof(int));
	glbGpuAllocator->_cudaMalloc((void **)&deviceValues, numKeys * sizeof(int));
	hostValues = (int *)malloc(numKeys * sizeof(int));
	if (!deviceKeys || !deviceValues || !hostValues) // ceck memory
		return NULL;
	hipMemcpy(deviceKeys, keys, numKeys * sizeof(int), hipMemcpyHostToDevice);
	// call kernel
	unsigned int blocks = ceil(1.0 * numKeys / THREADS_PER_BLOCK);
	kernel_get<<<blocks, THREADS_PER_BLOCK>>>(deviceKeys, deviceValues, numKeys, GPUHTable);
	hipDeviceSynchronize(); // wait
	cudaCheckError(); // ceck error
	hipMemcpy(hostValues, deviceValues, numKeys * sizeof(int), hipMemcpyDeviceToHost);
	glbGpuAllocator->_cudaFree(deviceValues); // free of memory
	glbGpuAllocator->_cudaFree(deviceKeys);
	return hostValues;
}
/**
 * Funtion get load factor
 */
float GpuHashTable::load_factor(void) { // calculate load factor
	if (GPUHTable.size == 0)
		return 0;
	return	GPUHTable.count * 1.0 / GPUHTable.size;
}

